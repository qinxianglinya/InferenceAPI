#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>
#include "kernel.h"
#include ""
#include "hip/device_functions.h"
#include <hip/hip_runtime.h>
#include "bboxUtils.h"


__global__ void sigmoid(const float *confData, float *confSigmoid, int *indexPtr, int dim, int batchSize)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= dim)
	{
		return;
	}
	for (int i = 0; i < batchSize; i++)
	{
		confSigmoid[i*dim + tid] = exp(confData[i*dim + tid]) / (1 + exp(confData[i*dim + tid]));
		indexPtr[i*dim + tid] = tid;
	}
}

__device__ float bboxSize(
	const float xmin, const float ymin, const float xmax, const float ymax)
{
	//printf("box size called...\n");
	if (xmax < xmin || ymax < ymin)
	{
		// If bbox is invalid (e.g. xmax < xmin or ymax < ymin), return 0.
		return 0;
	}
	else
	{
		float width = xmax - xmin;
		float height = ymax - ymin;

		return width * height;
	}
}

#define max(a,b) ( ((a)>(b)) ? (a):(b) )
#define min(a,b) ( ((a)>(b)) ? (b):(a) )

__device__ float intersectBbox(
	const float xmin1, const float ymin1, const float xmax1, const float ymax1,
	const float xmin2, const float ymin2, const float xmax2, const float ymax2)
{
	if (xmin2 > xmax1 || xmax2 < xmin1 || ymin2 > ymax1 || ymax2 < ymin1)
	{
		// Return [0, 0, 0, 0] if there is no intersection.
		return 0;
	}
	else
	{
		return bboxSize(max(xmin1, xmin2), max(ymin1, ymin2),
			min(xmax1, xmax2), min(ymax1, ymax2));
	}
}


__device__ float getIou(const float xmin1, const float ymin1, const float xmax1, const float ymax1,
	const float xmin2, const float ymin2, const float xmax2, const float ymax2)
{
	float intersect = intersectBbox(xmin1, ymin1, xmax1, ymax1, xmin2, ymin2, xmax2, ymax2);

	if (intersect > 0)
	{
		float bbox1_size = bboxSize(xmin1, ymin1, xmax1, ymax1);
		float bbox2_size = bboxSize(xmin2, ymin2, xmax2, ymax2);
		return intersect / (bbox1_size + bbox2_size - intersect);
	}
	else
	{
		return 0.;
	}
}

//suppressed:0��ʾ������1��ʾ����
__global__  void nms(const float* predictLoc, const int* index,
	bool *suppressed, float iouThreshold, const int dims, int nbLayer, int keepTopK, int batchSize)
{
	//dims����0��sizePerbatch��
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= dims)
	{
		return;
	}

	int sizePerbatch = nbLayer * keepTopK;

	for (int i = 0; i < batchSize; i++)
	{
		int s = index[i * sizePerbatch + tid];

		//�������
		float xmin = predictLoc[sizePerbatch * i * 4 + s * 4];
		float ymin = predictLoc[sizePerbatch * i * 4 + s * 4 + 1];
		float xmax = predictLoc[sizePerbatch * i * 4 + s * 4 + 2];
		float ymax = predictLoc[sizePerbatch * i * 4 + s * 4 + 3];

		if ((tid + 1) < sizePerbatch)
		{
			for (int j = tid + 1; j < sizePerbatch; j++)
			{
				//�Ƚ�iou,����������
				int d = index[i * sizePerbatch + j];
				float xmin1 = predictLoc[sizePerbatch * i * 4 + d * 4];
				float ymin1 = predictLoc[sizePerbatch * i * 4 + d * 4 + 1];
				float xmax1 = predictLoc[sizePerbatch * i * 4 + d * 4 + 2];
				float ymax1 = predictLoc[sizePerbatch * i * 4 + d * 4 + 3];
				float size2 = bboxSize(xmin1, ymin1, xmax1, ymax1);

				if (size2 <= 0)
				{
					suppressed[i * sizePerbatch * sizePerbatch + tid * sizePerbatch + j] = true;
				}
				else
				{
					float iou;
					iou = getIou(xmin, ymin, xmax, ymax, xmin1, ymin1, xmax1, ymax1);
					if (iou > iouThreshold)
					{
						suppressed[i * sizePerbatch * sizePerbatch + tid * sizePerbatch + j] = true;
					}
				}
			}
		}
	}
}




//priorNum:ÿ�����������ĸ���
__global__ void permuteData(const float *input, float *output, int num, int devideNum, int featureSize, int priorNum, int batchSize)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= num)
	{
		return;
	}
	int numPerbatch = num * devideNum * priorNum;

	//printf("before permute score: %f\n", input[tid]);
	//if (tid == 0)
	//{
	//	
	//}

	for (int s = 0; s < batchSize; s++)
	{
		for (int i = 0; i < priorNum; i++)
		{
			for (int j = 0; j < devideNum; j++)
			{
				output[s*numPerbatch + tid * priorNum*devideNum + i * devideNum + j] = input[s*numPerbatch + (i * devideNum*featureSize) + (j*featureSize) + tid];
				/*	if (tid == 0 && s == 1)
					{
						if (i == 0  && devideNum == 5)
						{
							printf("conf input%d:%f\n", tid, input[tid]);
							printf("input:%f\n", input[s*numPerbatch + (i * devideNum*featureSize) + (j*featureSize) + tid]);
							printf("output:%f\n", output[s*numPerbatch + tid * priorNum*devideNum + i * devideNum + j]);
						}
					}*/
			}
		}
	}

}

__global__ void permuteData2(const float *input, float *output, int num, int devideNum, int featureSize, int priorNum, int batchSize)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= num)
	{
		return;
	}
	int numPerbatch = num * devideNum * priorNum;

	//if (tid == 0)
	//{
	//	printf("before permute score: %f\n", input[tid]);
	//}
	//for (int s = 0; s < batchSize; s++)
	//{
	//	if (s == 1&&tid==0)
	//	{
	//		printf("%f\n", input[s*numPerbatch]);
	//	}
	//}

	for (int s = 0; s < batchSize; s++)
	{
		for (int i = 0; i < priorNum; i++)
		{
			for (int j = 0; j < devideNum; j++)
			{
				output[s*numPerbatch + tid * priorNum*devideNum + i * devideNum + j] = input[s*numPerbatch + (i * devideNum*featureSize) + (j*featureSize) + tid];
				/*			if (s == 1 && i == 0)
							{
								printf("output:%f\n", output[s*numPerbatch + tid * priorNum*devideNum + i * devideNum + j]);
							}*/
							/*	if (tid == 0 && s == 1)
								{
									if (i == 0  && devideNum == 5)
									{
										printf("conf input%d:%f\n", tid, input[tid]);
										printf("input:%f\n", input[s*numPerbatch + (i * devideNum*featureSize) + (j*featureSize) + tid]);
										printf("output:%f\n", output[s*numPerbatch + tid * priorNum*devideNum + i * devideNum + j]);
									}
								}*/
			}
		}
	}

}


__global__ void setOffset(int *offset, int dims, int batchSize)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid > 0)
	{
		return;
	}
	offset[0] = 0;
	for (int i = 1; i < batchSize + 1; i++)
	{
		offset[i] = i * dims;
	}
	/*offset[tid] = 0;
	offset[tid + 1] = dims;*/
}

__global__ void getTopkNum(const float *inputScore, const int *inputIndex, float *outputScore, int *outputIndex,
	float threshold, const int dims, int *anchorIndex, int *classIndex, const int classNum, int batchSize, int totalScoreNum)
{
	//dimsΪkeeptopk
	//totalScoreNum:featureSize * 9 * numCls
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= dims)
	{
		return;
	}
	//printf("inputIndex:%d\n", inputIndex[tid]);
	//outputScore[tid] = inputScore[tid];
	//outputIndex[tid] = inputIndex[tid];
	for (int i = 0; i < batchSize; i++)
	{
		if (inputScore[i*totalScoreNum + tid] >= threshold)
		{
			//printf("%f\n", inputScore[i*totalScoreNum + tid]);
			outputScore[i*dims + tid] = inputScore[i*totalScoreNum + tid];
			outputIndex[i*dims + tid] = inputIndex[i*totalScoreNum + tid];
			//upThreshold[tid] = 1;
			anchorIndex[i*dims + tid] = outputIndex[i*dims + tid] / classNum;//ê������

			//printf("anchorindex:%d\n", anchorIndex[tid]);
			classIndex[i*dims + tid] = outputIndex[i*dims + tid] % classNum;//�����
			//if(i==1)
			//printf("%d\n", anchorIndex[i*dims + tid]);
	/*		if (i == 1)
			{
				printf("%d\n", anchorIndex[i*dims + tid]);
			}*/
		}
		else
		{
			//upThreshold[tid] = 0;
			outputScore[i*dims + tid] = 0.0f;
			outputIndex[i*dims + tid] = -1;
			anchorIndex[i*dims + tid] = -1;
			classIndex[i*dims + tid] = -1;
		}
		/*	if (i == 1)
			{
				printf("anchorIndex[i*dims + tid]:%d\n", anchorIndex[i*dims + tid]);
			}*/
	}


}

//��ÿһ�����������������
__global__ void concatArray(const float* beforeBox, const float* beforeScore, const int* beforeClass, float* afterBox, float* afterScore, int* afterClass,
	int layerIndex, int dims, int batchSize, int keepK, int layerNum, int keepTopK)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= dims)
	{
		return;
	}
	for (int i = 0; i < batchSize; i++)
	{
		if (tid < keepK)
		{
			//�ں�box
			afterBox[i*dims * 4 * layerNum + layerIndex * 4 * keepTopK + tid * 4] = beforeBox[i*keepK * 4 + tid * 4];
			afterBox[i*dims * 4 * layerNum + layerIndex * 4 * keepTopK + tid * 4 + 1] = beforeBox[i*keepK * 4 + tid * 4 + 1];
			afterBox[i*dims * 4 * layerNum + layerIndex * 4 * keepTopK + tid * 4 + 2] = beforeBox[i*keepK * 4 + tid * 4 + 2];
			afterBox[i*dims * 4 * layerNum + layerIndex * 4 * keepTopK + tid * 4 + 3] = beforeBox[i*keepK * 4 + tid * 4 + 3];
			//�ں�score
			afterScore[i* layerNum * keepTopK + layerIndex * keepTopK + tid] = beforeScore[i*keepK + tid];
			//if (afterScore[i* layerNum * keepTopK + layerIndex * keepTopK + tid] != 0)
			//{
			//	printf("concat score:!!!%f\n", afterScore[i* layerNum * keepTopK + layerIndex * keepTopK + tid]);
			//}
			//�ں�class
			afterClass[i* layerNum * keepTopK + layerIndex * keepTopK + tid] = beforeClass[i*keepK + tid];
		}
		else
		{
			//�ں�box
			//printf("tid con:%d\n", tid);
			afterBox[i*dims * 4 * layerNum + layerIndex * 4 * keepTopK + tid * 4] = 0;
			afterBox[i*dims * 4 * layerNum + layerIndex * 4 * keepTopK + tid * 4 + 1] = 0;
			afterBox[i*dims * 4 * layerNum + layerIndex * 4 * keepTopK + tid * 4 + 2] = 0;
			afterBox[i*dims * 4 * layerNum + layerIndex * 4 * keepTopK + tid * 4 + 3] = 0;
			//�ں�score
			afterScore[i*layerNum * keepTopK + layerIndex * keepTopK + tid] = 0;
			//�ں�class
			afterClass[i*layerNum * keepTopK + layerIndex * keepTopK + tid] = (-1);
		}

		//if (i == 1 && afterScore[i*layerNum * keepTopK + layerIndex * keepTopK + tid] != 0&&layerIndex == 2)
		//{
		//	//printf("layer index:%d\n", layerIndex);
		//	printf("after box:%d, %f, %f, %f, %f, %f, %d\n", tid, afterBox[i*dims * 4 * layerNum + layerIndex * 4 * keepTopK + tid * 4],
		//		afterBox[i*dims * 4 * layerNum + layerIndex * 4 * keepTopK + tid * 4 + 1], afterBox[i*dims * 4 * layerNum + layerIndex * 4 * keepTopK + tid * 4 + 2],
		//		afterBox[i*dims * 4 * layerNum + layerIndex * 4 * keepTopK + tid * 4 + 3], afterScore[i*layerNum * keepTopK + layerIndex * keepTopK + tid],
		//		afterClass[i*layerNum * keepTopK + layerIndex * keepTopK + tid]);
		//	printf("before box:%d, %f, %f, %f, %f, %f, %d\n", tid, beforeBox[i*keepK * 4 + tid * 4], beforeBox[i*keepK * 4 + tid * 4 + 1], beforeBox[i*keepK * 4 + tid * 4 + 2],
		//		beforeBox[i*keepK * 4 + tid * 4 + 3], beforeScore[i*keepK + tid], beforeClass[i*keepK + tid]);
		//}

	}
}

__global__ void decode(const float *anchor, const float *locData, float *predictBox, int dims, float scaleClamp, int batchSize)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= dims)
	{
		return;
	}
	if (locData[tid] != 0)
	{

	}
	for (int i = 0; i < batchSize; i++)
	{
		//if (anchor[i*dims * 4 +tid * 4] == 0.0f&&locData[i*dims * 4 + tid * 4] == 0.0f&&
		//	anchor[i*dims * 4 + tid * 4+1] == 0.0f&&locData[i*dims * 4 + tid * 4+1] == 0.0f&&
		//	anchor[i*dims * 4 + tid * 4+2] == 0.0f&&locData[i*dims * 4 + tid * 4+2] == 0.0f&&
		//	anchor[i*dims * 4 + tid * 4+3] == 0.0f&&locData[i*dims * 4 + tid * 4+3] == 0.0f)
		//{
		//	predictBox[i*dims*4 + tid * 4] = 0.0f;
		//	predictBox[i*dims*4 + tid * 4 + 1] = 0.0f;
		//	predictBox[i*dims*4 + tid * 4 + 2] = 0.0f;
		//	predictBox[i*dims*4 + tid * 4 + 3] = 0.0f;
		//	return;
		//}

		//���н������
		//torch.clamp:���������Сֵ
		float anchorW = anchor[i*dims * 4 + tid * 4 + 2] - anchor[i*dims * 4 + tid * 4];
		float anchorH = anchor[i*dims * 4 + tid * 4 + 3] - anchor[i*dims * 4 + tid * 4 + 1];
		float anchorCx = anchor[i*dims * 4 + tid * 4] + 0.5 * anchorW;
		float anchorCy = anchor[i*dims * 4 + tid * 4 + 1] + 0.5 * anchorH;

		float dx = locData[i*dims * 4 + tid * 4];
		float dy = locData[i*dims * 4 + tid * 4 + 1];
		float dw = locData[i*dims * 4 + tid * 4 + 2];
		float dh = locData[i*dims * 4 + tid * 4 + 3];
		/*	if (locData[tid] != 0)
			{
				printf("decode input:%f, %f, %f, %f\n", locData[tid * 4], locData[tid * 4 + 1], locData[tid * 4 + 1], locData[tid * 4 + 1]);
				printf("%f, %f, %f, %f, %f, %f, %f, %f\n", anchorW, anchorH, anchorCx, anchorCy, dx, dy, dw, dh);

			}*/

		if (dw > scaleClamp)
		{
			dw = scaleClamp;
		}
		if (dh > scaleClamp)
		{
			dh = scaleClamp;
		}

		float preCx = dx * anchorW + anchorCx;
		float preCy = dy * anchorH + anchorCy;
		float preW = anchorW * exp(dw);
		float preH = anchorH * exp(dh);

		predictBox[i*dims * 4 + tid * 4] = preCx - 0.5 * preW;
		predictBox[i*dims * 4 + tid * 4 + 1] = preCy - 0.5 * preH;
		predictBox[i*dims * 4 + tid * 4 + 2] = preCx + 0.5 * preW;
		predictBox[i*dims * 4 + tid * 4 + 3] = preCy + 0.5 * preH;
		/*	if (i == 1&& predictBox[i*dims * 4 + tid * 4]!=0)
			{
				printf("predict box: %d,  %f, %f,  %f,  %f  \n", i*dims * 4 + tid * 4, predictBox[i*dims * 4 + tid * 4], predictBox[i*dims * 4 + tid * 4 + 1],
					predictBox[i*dims * 4 + tid * 4 + 2], predictBox[i*dims * 4 + tid * 4 + 3]);
			}*/
	}


}

__global__ void getBoxAndAnchor(const int *anchorIndex, float *inputAnchor, float *inputLoc, float *outputAnchor, float *outputLoc, int dims, int batchSize, int boxStart)
{
	//dims : keeptopk
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= dims)
	{
		return;
	}

	for (int i = 0; i < batchSize; i++)
	{
		//if (i == 1 && tid < 10)
		//{
		//	printf("%f\n", inputLoc[tid]);
		//}
		if (anchorIndex[i*dims + tid] == (-1))
		{
			outputAnchor[i*dims * 4 + tid * 4] = 0.0f;
			outputAnchor[i*dims * 4 + tid * 4 + 1] = 0.0f;
			outputAnchor[i*dims * 4 + tid * 4 + 2] = 0.0f;
			outputAnchor[i*dims * 4 + tid * 4 + 3] = 0.0f;
			outputLoc[i*dims * 4 + tid * 4] = 0.0f;
			outputLoc[i*dims * 4 + tid * 4 + 1] = 0.0f;
			outputLoc[i*dims * 4 + tid * 4 + 2] = 0.0f;
			outputLoc[i*dims * 4 + tid * 4 + 3] = 0.0f;
		}
		else
		{
			//printf("%d\n", (anchorIndex[tid]));
			outputAnchor[i*dims * 4 + tid * 4] = inputAnchor[(anchorIndex[i*dims + tid]) * 4];
			outputAnchor[i*dims * 4 + tid * 4 + 1] = inputAnchor[(anchorIndex[i*dims + tid]) * 4 + 1];
			outputAnchor[i*dims * 4 + tid * 4 + 2] = inputAnchor[(anchorIndex[i*dims + tid]) * 4 + 2];
			outputAnchor[i*dims * 4 + tid * 4 + 3] = inputAnchor[(anchorIndex[i*dims + tid]) * 4 + 3];



			outputLoc[i*dims * 4 + tid * 4] = inputLoc[boxStart*i + (anchorIndex[i*dims + tid]) * 4];
			outputLoc[i*dims * 4 + tid * 4 + 1] = inputLoc[boxStart*i + (anchorIndex[i*dims + tid]) * 4 + 1];
			outputLoc[i*dims * 4 + tid * 4 + 2] = inputLoc[boxStart*i + (anchorIndex[i*dims + tid]) * 4 + 2];
			outputLoc[i*dims * 4 + tid * 4 + 3] = inputLoc[boxStart*i + (anchorIndex[i*dims + tid]) * 4 + 3];

			//if (i == 1)
			//{
			///*	printf("%f,%f,%f,%f\n", outputLoc[i*dims * 4 + tid * 4],
			//		outputLoc[i*dims * 4 + tid * 4 + 1],
			//		outputLoc[i*dims * 4 + tid * 4 + 2],
			//		outputLoc[i*dims * 4 + tid * 4 + 3]);*/
			//	printf("%d, %f\n", anchorIndex[i*dims + tid], inputLoc[(anchorIndex[i*dims + tid]) * 4]);
			//}

		/*	if (tid < 10)
			{
				printf("input anchor:%f, %f, %f, %f\n", inputAnchor[tid * 4], inputAnchor[tid * 4 + 1], inputAnchor[tid * 4 + 2], inputAnchor[tid * 4 + 3]);
				printf("outputAnchor:%f, %f, %f, %f\n", outputAnchor[i*dims * 4 + tid * 4], outputAnchor[i*dims * 4 + tid * 4 + 1],
					outputAnchor[i*dims * 4 + tid * 4 + 2], outputAnchor[i*dims * 4 + tid * 4 + 3]);
			}*/
			//printf("outputAnchor:%f, %f, %f, %f\n", outputAnchor[i*dims * 4 + tid * 4], outputAnchor[i*dims * 4 + tid * 4 + 1],
			//	outputAnchor[i*dims * 4 + tid * 4 + 2], outputAnchor[i*dims * 4 + tid * 4 + 3]);

			/*printf("output loc%d,%f\n", anchorIndex[tid], outputLoc[i*dims * 4 + tid * 4]);
			printf("output loc%d,%f\n", anchorIndex[tid], outputLoc[i*dims * 4 + tid * 4 + 1]);
			printf("output loc%d,%f\n", anchorIndex[tid], outputLoc[i*dims * 4 + tid * 4 + 2]);
			printf("output loc%d,%f\n", anchorIndex[tid], outputLoc[i*dims * 4 + tid * 4 + 3]);*/
		}
	}

}


__global__ void getBoxForNms(const float* box, const int* classIndex, const float* sortedBox, float* boxForNms, int batchSize, int layerNum, int dims, int* indexPtr, int* suppressBox, int keepTopK)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= dims)
	{
		return;
	}
	//printf("getboxfornms\n");
	for (int i = 0; i < batchSize; i++)
	{
		float maxbox = sortedBox[i * keepTopK * 4 * layerNum] + 1;
		/*	if (tid == 0)
			{
				printf("maxbox:%f\n", maxbox);
			}*/

		indexPtr[i * keepTopK * layerNum + tid] = tid;
		suppressBox[i * keepTopK * layerNum + tid] = 0;
		//float offset = maxbox * 
	/*	if (tid == 2002)
		{
			printf("2002:%f, %f, %f, %f\n", box[2002 * 4], box[2002 * 4 + 1], box[2002 * 4 + 2], box[2002 * 4 + 3]);
		}*/
		if (box[i * keepTopK * layerNum * 4 + tid * 4] == 0 && box[i * keepTopK * layerNum * 4 + tid * 4 + 3] == 0)
		{
			boxForNms[i * keepTopK * layerNum * 4 + tid * 4] = 0.0f;
			boxForNms[i * keepTopK * layerNum * 4 + tid * 4 + 1] = 0.0f;
			boxForNms[i * keepTopK * layerNum * 4 + tid * 4 + 2] = 0.0f;
			boxForNms[i * keepTopK * layerNum * 4 + tid * 4 + 3] = 0.0f;
		}
		else
		{
			//printf("%d\n", classIndex[2002]);
			//printf("class index%d:%d\n", i * keepTopK * layerNum + tid,classIndex[i * keepTopK * layerNum + tid]);

			//printf("maxbox:%d\n", classIndex[i * keepTopK * layerNum + tid]);
			boxForNms[i * keepTopK * layerNum * 4 + tid * 4] = box[i * keepTopK * layerNum * 4 + tid * 4] + maxbox * classIndex[i * keepTopK * layerNum + tid];
			boxForNms[i * keepTopK * layerNum * 4 + tid * 4 + 1] = box[i * keepTopK * layerNum * 4 + tid * 4 + 1] + maxbox * classIndex[i * keepTopK * layerNum + tid];
			boxForNms[i * keepTopK * layerNum * 4 + tid * 4 + 2] = box[i * keepTopK * layerNum * 4 + tid * 4 + 2] + maxbox * classIndex[i * keepTopK * layerNum + tid];
			boxForNms[i * keepTopK * layerNum * 4 + tid * 4 + 3] = box[i * keepTopK * layerNum * 4 + tid * 4 + 3] + maxbox * classIndex[i * keepTopK * layerNum + tid];
			//debug 0118 -- boxforNms 
			//offset���������*��ǰbox���ֵ��Ϊoffset
		/*	printf("%f,%f,%f,%f\n", boxForNms[i * keepTopK * layerNum * 4 + tid * 4],
				boxForNms[i * keepTopK * layerNum * 4 + tid * 4 + 1],
				boxForNms[i * keepTopK * layerNum * 4 + tid * 4 + 2],
				boxForNms[i * keepTopK * layerNum * 4 + tid * 4 + 3]);*/
		}
	}
}

__global__ void getResultAfterNms(const float* box, const float* score, const int* classIndex, const int* indexSorted, const bool* suppressedIndex, float* scoreAfterNms,
	float* boxAfterNms, int* classIndexAfterNms, int topK, int batchSize, int layerNum, int keepTopK)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	//if(tid>=topK)
	if (tid >= layerNum * keepTopK)
	{
		return;
	}
	int numPerBatch = layerNum * keepTopK;
	for (int i = 0; i < batchSize; i++)
	{
		//printf("tid:%d\n", tid);
		int k = indexSorted[i*numPerBatch + tid];
		if (suppressedIndex[i*numPerBatch + tid] == false)
		{
			boxAfterNms[i*numPerBatch * 4 + tid * 4] = box[i*numPerBatch * 4 + k * 4];
			boxAfterNms[i*numPerBatch * 4 + tid * 4 + 1] = box[i*numPerBatch * 4 + k * 4 + 1];
			boxAfterNms[i*numPerBatch * 4 + tid * 4 + 2] = box[i*numPerBatch * 4 + k * 4 + 2];
			boxAfterNms[i*numPerBatch * 4 + tid * 4 + 3] = box[i*numPerBatch * 4 + k * 4 + 3];
			scoreAfterNms[i*numPerBatch + tid] = score[i*numPerBatch + k];
			classIndexAfterNms[i*numPerBatch + tid] = classIndex[i*numPerBatch + k];//��ȷ��
		}
		else
		{
			//int k = indexSorted[i*numPerBatch + tid];
			boxAfterNms[i*numPerBatch * 4 + tid * 4] = 0.0f;
			boxAfterNms[i*numPerBatch * 4 + tid * 4 + 1] = 0.0f;
			boxAfterNms[i*numPerBatch * 4 + tid * 4 + 2] = 0.0f;
			boxAfterNms[i*numPerBatch * 4 + tid * 4 + 3] = 0.0f;
			scoreAfterNms[i*numPerBatch + tid] = 0.0f;
			//printf("getresult2:%f\n", scoreAfterNms[i*numPerBatch + tid]);
			classIndexAfterNms[i*numPerBatch + tid] = -1;
		}
	}

}
//
__device__ float clamp(float data, int limitMax)
{
	if (data < 0)
	{
		return 0.0f;
	}
	else if (data > limitMax)
	{
		return limitMax * 1.0;
	}
	else
	{
		return data;
	}
}

//__global__ void scaleAndClip(const float* box, float* boxAfterScale, int srcW, int srcH, float scaleW, float scaleH, int topK, int batchSize, int layerNum, int keepTopK)
//{
//	int tid = blockIdx.x * blockDim.x + threadIdx.x;
//	topK = layerNum * keepTopK;
//	if (tid >= topK)
//	{
//		return;
//	}
//	for (int i = 0; i < batchSize; i++)
//	{
//		//if (box[i* topK * 4 + tid * 4] != 0)
//		//{
//		//	//printf("%f, %f, %f, %f\n", box[i* topK * 4 + tid * 4], box[i* topK * 4 + tid * 4 + 1], box[i* topK * 4 + tid * 4 + 2], box[i* topK * 4 + tid * 4 + 3]);
//		//}
//		if (box[i* topK * 4 + tid*4] != 0 && box[i* topK * 4 + tid * 4 + 1] != 0 &&
//			box[i* topK * 4 + tid*4 + 2] != 0 && box[i* topK * 4 + tid*4 + 3] != 0)
//		{
//			float xmin = box[i* topK * 4 + tid*4] * scaleW;
//			float xmax = box[i*topK * 4 + tid * 4 + 2] * scaleW;
//			float ymin = box[i*topK * 4 + tid * 4 + 1] * scaleH;
//			float ymax = box[i*topK * 4 + tid * 4 + 3] * scaleH;
//			//printf("%f, %f, %f, %f\n", xmin, ymin, xmax, ymax);
//			xmin = clamp(xmin, srcW);
//			xmax = clamp(xmax, srcW);
//			ymin = clamp(ymin, srcH);
//			ymax = clamp(ymax, srcH);
//			/*printf("%f, %f, %f, %f\n", xmin, ymin, xmax, ymax);*/
//
//			boxAfterScale[i * topK * 4 + tid * 4] = xmin;
//			boxAfterScale[i * topK * 4 + tid * 4 + 1] = ymin;
//			boxAfterScale[i * topK * 4 + tid * 4 + 2] = xmax;
//			boxAfterScale[i * topK * 4 + tid * 4 + 3] = ymax;
//		}
//		else
//		{
//			boxAfterScale[i * topK * 4 + tid * 4] = 0.0f;
//			boxAfterScale[i * topK * 4 + tid * 4 + 1] = 0.0f;
//			boxAfterScale[i * topK * 4 + tid * 4 + 2] = 0.0f;
//			boxAfterScale[i * topK * 4 + tid * 4 + 3] = 0.0f;
//		}
//	}
//
//
//}

//debug permute
__global__ void test(float* input, const int dims)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= dims)
	{
		return;
	}
	if (tid == 0)
	{
		printf("%f\n", input[tid]);
	}
	//if (tid == 6163 * 4)
	//{
	//	printf("%f\n", input[6163 * 4]);
	//}
	//if (tid >= (dims / 2)&&tid<=(dims/2)+100)
	//{
	//	printf("%d, %f\n", tid, input[tid]);
	//		//input[tid*4], input[tid*4+1], input[tid*4+2], input[tid*4+3]);
	//}
}
//debug index
__global__ void test1(float* input, int dims)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= dims)
	{
		return;
	}
	if (input[tid * 4] != 0)
	{
		printf("%f, %f, %f, %f\n", input[tid * 4], input[tid * 4 + 1], input[tid * 4 + 2], input[tid * 4 + 3]);
	}
	//if (tid >= (dims/2) && tid < (dims / 2)+30)
	//{
	//	printf("%d\n", input[tid]);
	//}
}

void* detectionInferenceTorch(hipStream_t stream, int batchSize, void *workspace, const void *anchor, const void *confData, const void *locData, void *boxPtr,
	void *scorePtr, void *classPtr, int layerIndex, int featureSize, int priorNum, int classNum, int keepK, int layerNum, float scoreThreshold, int keepTopK)
{


	const int dims = featureSize * priorNum * classNum;
	const int BS = 128;
	const int GS = (dims + BS - 1) / BS;

	//1���ı�Ԥ�����Ŷȵ�ά��
	void *permuteConf = workspace;
	size_t permuteConfSize = floatSize(batchSize, featureSize * priorNum * classNum);

	const int GS1 = (featureSize + BS - 1) / BS;
	permuteData << <GS1, BS >> > ((float*)confData, (float*)permuteConf, dims / priorNum / classNum, classNum, featureSize, priorNum, batchSize);
	//std::cout << "--------------------------------" << std::endl;

	//2���ı�Ԥ��ƫ��ֵ��ά��
	void *permuteLoc = nextWorkspacePtr((int8_t*)permuteConf, permuteConfSize);
	size_t permuteLoc1Size = floatSize(batchSize, featureSize * priorNum * 4);

	const int dimss = featureSize * priorNum * 4;
	const int GSS = (featureSize + BS - 1) / BS;
	permuteData2 << <GSS, BS >> > ((float*)locData, (float*)permuteLoc, dimss / priorNum / 4, 4, featureSize, priorNum, batchSize);



	//3����Ԥ�����ŶȽ���sigmoid
	void *sigmoidConf = nextWorkspacePtr((int8_t*)permuteLoc, permuteLoc1Size);
	size_t sigmoidSize = floatSize(batchSize, featureSize * priorNum * classNum);

	//4��indexPtr������Ŷȶ�Ӧ������
	//sigmoidConf������Ŷ�
	void *indexPtr = nextWorkspacePtr((int8_t*)sigmoidConf, sigmoidSize);
	size_t indexSize = intSize(batchSize, featureSize * priorNum * classNum);
	sigmoid << <GS, BS >> > ((float*)permuteConf, (float *)sigmoidConf, (int*)indexPtr, featureSize * priorNum * classNum, batchSize);



	//5�������ŶȽ�������
	void     *d_temp_storage = NULL;
	size_t   temp_storage_bytes = 0;
	const int num_items = batchSize * featureSize * priorNum * classNum;
	const int num_segments = batchSize;

	void *d_offsets = nextWorkspacePtr((int8_t*)indexPtr, indexSize);
	size_t offsetSize = (num_segments + 1) * sizeof(int);

	setOffset << <1, 1 >> > ((int*)d_offsets, featureSize * priorNum * classNum, batchSize);


	//d_scoreSorted��ű������������Ŷ�
	void *d_scoreSorted = nextWorkspacePtr((int8_t*)d_offsets, offsetSize);
	size_t scoreSortedSize = floatSize(batchSize, featureSize * priorNum * classNum);
	//indexSorted�����������Ŷ���ԭʼ�����е�����
	void *indexSorted = nextWorkspacePtr((int8_t*)d_scoreSorted, scoreSortedSize);
	size_t indexSortedSize = intSize(batchSize, featureSize * priorNum * classNum);

	hipcub::DeviceSegmentedRadixSort::SortPairsDescending(d_temp_storage, temp_storage_bytes, (const float*)sigmoidConf, (float*)d_scoreSorted, (const int*)indexPtr, (int *)indexSorted,
		num_items, num_segments, (const int*)d_offsets, (const int*)d_offsets + 1, 0, sizeof(float) * 8,
		stream);
	hipMalloc(&d_temp_storage, temp_storage_bytes);
	hipcub::DeviceSegmentedRadixSort::SortPairsDescending(d_temp_storage, temp_storage_bytes, (const float*)sigmoidConf, (float*)d_scoreSorted, (const int*)indexPtr, (int *)indexSorted,
		num_items, num_segments, (const int*)d_offsets, (const int*)d_offsets + 1, 0, sizeof(float) * 8,
		stream);

	//test << <2, 1 >> > ((float*)confData, 1);
	//printf("-----------\n");

	//const int testdims = featureSize * priorNum * 2;
	//const int testBS = 128;
	//const int testGS = (testdims + testBS - 1) / testBS;
	//test << <testGS, testBS >> > ((float*)d_scoreSorted, testdims);

	//6����ȡǰkeepK���÷֡�����
	//predictProb����ǰkeepK���÷�
	void *predictProb = nextWorkspacePtr((int8_t*)indexSorted, indexSortedSize);
	size_t predictProbSize = floatSize(batchSize, keepK);
	//predictIndex����ǰkeepK���÷�����Ӧԭʼ��������
	void *predictIndex = nextWorkspacePtr((int8_t*)predictProb, predictProbSize);
	size_t topkIndex = intSize(batchSize, keepK);
	//anchorIndex����ǰkeepK���÷�����Ӧê�������
	void *anchorIndex = nextWorkspacePtr((int8_t*)predictIndex, topkIndex);
	size_t anchorIndexSize = intSize(batchSize, keepK);
	//classIndex����ǰkeepK�÷ֵ�Ԥ����Ԥ�����
	void *classIndex = nextWorkspacePtr((int8_t*)anchorIndex, anchorIndexSize);
	size_t classIndexSize = intSize(batchSize, keepK);

	const int GS2 = (1000 + BS - 1) / BS;
	getTopkNum << <GS2, BS >> > ((const float*)d_scoreSorted, (const int*)indexSorted, (float*)predictProb, (int*)predictIndex, (float)scoreThreshold, keepK,
		(int*)anchorIndex, (int*)classIndex, classNum, batchSize, featureSize*priorNum*classNum);

	//7����ȡԤ��ƫ��ֵ�Լ���Ӧê��
	void *boxReg = nextWorkspacePtr((int8_t*)classIndex, classIndexSize);
	size_t boxRegSize = floatSize(batchSize, keepK * 4);

	void *anchors_i = nextWorkspacePtr((int8_t*)boxReg, boxRegSize);
	size_t anchorsSize = floatSize(batchSize, keepK * 4);



	//debug index
	const int testdimsindex = 80 * 80 * 9 * 2;
	const int testBsIndex = 128;
	const int testGsIndex = (testdimsindex + testBsIndex - 1) / testBsIndex;
	//test1 << <testGsIndex, testBsIndex >> > ((int*)indexSorted, testdimsindex);
	//anchorIndex��ÿһ��batch���������Ǵ�0��ʼ������batch��Ӧ��box������Ҫ����ǰ��batch��box����
	getBoxAndAnchor << <GS2, BS >> > ((const int*)anchorIndex, (float*)anchor, (float*)permuteLoc, (float*)anchors_i, (float*)boxReg, keepK, batchSize, featureSize*priorNum * 4);

	void *afterDecode = nextWorkspacePtr((int8_t*)anchors_i, anchorsSize);
	size_t afterDecodeSize = floatSize(batchSize, keepK * 4);

	void* next = nextWorkspacePtr((int8_t*)afterDecode, afterDecodeSize);

	//8�����룬�õ�Ԥ���
	decode << <GS2, BS >> > ((const float*)anchors_i, (const float*)boxReg, (float*)afterDecode, keepK, 4.1352, batchSize);

	//9�����Ӹ�������
	concatArray << <GS2, BS >> > ((const float*)afterDecode, (const float*)predictProb, (const int*)classIndex, (float*)boxPtr, (float*)scorePtr, (int*)classPtr, layerIndex, keepTopK, batchSize, keepK, layerNum, keepTopK);
	/*printf("%f\n", confData);*/
	hipFree(d_temp_storage);
	return next;
}

__global__ void init_suppress(bool* suppress, bool* suppress_1d, int dims, int batchSize, int layerNum, int keepTopK)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= dims)
	{
		return;
	}
	for (int i = 0; i < batchSize; i++)
	{
		suppress[i * dims + tid] = false;
		if (tid < (layerNum * keepTopK))
		{
			suppress_1d[i * layerNum * keepTopK + tid] = false;
		}
	}
}

__global__ void get_suppress(const bool* suppressIn, bool* suppressOut, int dims, int batchSize)
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if (tid >= dims)
	{
		return;
	}
	for (int i = 0; i < batchSize; i++)
	{
		for (int j = 0; j < dims; j++)
		{
			if (suppressIn[i * dims * dims + j * dims + tid] == true)
			{
				//printf("111\n");
				suppressOut[i * dims + tid] = true;
				break;
			}
		}
	}
}

pluginStatus_t batchNms(hipStream_t stream, int batchSize, void *workspace, const void *box, const void *score, const void *classIndex, float iouthreshold, int classNum, int layerNum, int topK,
	int srcW, int srcH, int tarW, int tarH, float* outLoc, float* outConf, int* outClass, int keepTopK)
{
	//1.�����ҳ�box���ֵ
	void     *d_temp_storage = NULL;
	void     *d_temp = NULL;
	size_t   temp_storage_bytes = 0;
	int num_segments = batchSize;

	void *sortedBox = workspace;
	size_t sortedBoxSize = predictDataSize(batchSize, layerNum * keepTopK * 4);

	void *offsets = nextWorkspacePtr((int8_t*)sortedBox, sortedBoxSize);
	size_t offsetSize = (num_segments + 1) * sizeof(int);
	//setoffset����������
	setOffset << <1, 1 >> > ((int*)offsets, keepTopK * 4 * layerNum, batchSize);

	hipcub::DeviceSegmentedRadixSort::SortKeysDescending(d_temp_storage, temp_storage_bytes, (const float*)box, (float*)sortedBox, batchSize * keepTopK * 4 * layerNum,
		num_segments, (const int*)offsets, (const int*)offsets + 1, 0, sizeof(float) * 8, stream);
	// Allocate temporary storage
	hipMalloc(&d_temp_storage, temp_storage_bytes);
	// Run sorting operation
	hipcub::DeviceSegmentedRadixSort::SortKeysDescending(d_temp_storage, temp_storage_bytes, (const float*)box, (float*)sortedBox, batchSize * keepTopK * 4 * layerNum,
		num_segments, (const int*)offsets, (const int*)offsets + 1, 0, sizeof(float) * 8, stream);

	void *boxForNmsPtr = nextWorkspacePtr((int8_t*)offsets, offsetSize);
	size_t boxForNmsSize = predictDataSize(batchSize, layerNum * keepTopK * 4);

	//�����ƫ��ֵ���box
	//int sizePerBatch = layerNum * 1000;
	void *indexPtr = nextWorkspacePtr((int8_t*)boxForNmsPtr, boxForNmsSize);
	size_t indexPtrSize = indexDataSize(batchSize, layerNum * keepTopK);

	void *suppressBox = nextWorkspacePtr((int8_t*)indexPtr, indexPtrSize);
	size_t suppressBoxSize = boolDataSize(batchSize, layerNum * keepTopK * layerNum * keepTopK);

	void *suppress_1d = nextWorkspacePtr((int8_t*)suppressBox, suppressBoxSize);
	size_t suppress_1d_size = boolDataSize(batchSize, layerNum * keepTopK);

	//����nms��ǰ100��Ԥ������Ϣ
	void *locAfterNms = nextWorkspacePtr((int8_t*)suppress_1d, suppress_1d_size);
	size_t locAfterNmsSize = predictDataSize(batchSize, layerNum*keepTopK * 4);

	const int BS = 128;
	const int GS = ((1000 * layerNum) + BS - 1) / BS;
	getBoxForNms << <GS, BS >> > ((float*)box, (int*)classIndex, (float*)sortedBox, (float*)boxForNmsPtr, batchSize, layerNum, layerNum * keepTopK, (int*)indexPtr, (int*)suppressBox, keepTopK);

	//nms
	//nms step1:�������򣬷���������box����
	void* offsetScore = nextWorkspacePtr((int8_t*)locAfterNms, locAfterNmsSize);
	size_t offsetScoreSize = (num_segments + 1) * sizeof(int);
	setOffset << <1, 1 >> > ((int*)offsetScore, keepTopK * layerNum, batchSize);


	void *sortedScore = nextWorkspacePtr((int8_t*)offsetScore, offsetScoreSize);
	size_t sortedScoreSize = sigmoidDataSize(batchSize, layerNum * keepTopK);

	void *indexSorted = nextWorkspacePtr((int8_t*)sortedScore, sortedScoreSize);
	size_t indexSortedSize = indexDataSize(batchSize, layerNum * keepTopK);
	//std::cout << "sorted start" << std::endl;
	hipcub::DeviceSegmentedRadixSort::SortPairsDescending(d_temp, temp_storage_bytes, (const float*)score, (float*)sortedScore, (const int*)indexPtr, (int *)indexSorted,
		batchSize * keepTopK * layerNum, num_segments, (const int*)offsetScore, (const int*)offsetScore + 1, 0, sizeof(float) * 8,
		stream);

	// Allocate temporary storage
	hipMalloc(&d_temp, temp_storage_bytes);

	//// Run sorting operation
	hipcub::DeviceSegmentedRadixSort::SortPairsDescending(d_temp, temp_storage_bytes, (const float*)score, (float*)sortedScore, (const int*)indexPtr, (int *)indexSorted,
		batchSize * keepTopK * layerNum, num_segments, (const int*)offsetScore, (const int*)offsetScore + 1, 0, sizeof(float) * 8,
		stream);

	int BS_INIT_SUPPRESS = 128;
	int GS_INIT_SUPPRESS = (layerNum * keepTopK * layerNum * keepTopK + BS_INIT_SUPPRESS - 1) / BS_INIT_SUPPRESS;
	init_suppress << <GS_INIT_SUPPRESS, BS_INIT_SUPPRESS >> > ((bool*)suppressBox, (bool*)suppress_1d, layerNum * keepTopK * layerNum * keepTopK, batchSize, layerNum, keepTopK);

	int GS_NMS = (layerNum * keepTopK + BS_INIT_SUPPRESS - 1) / BS_INIT_SUPPRESS;
	nms << <GS_NMS, BS_INIT_SUPPRESS >> > ((float*)boxForNmsPtr, (int*)indexSorted, (bool*)suppressBox, iouthreshold, layerNum * keepTopK, layerNum, keepTopK, batchSize);

	get_suppress << <GS_NMS, BS_INIT_SUPPRESS >> > ((bool*)suppressBox, (bool*)suppress_1d, (layerNum * keepTopK), batchSize);

	getResultAfterNms << <10, 800 >> > ((float*)box, (float*)score, (int*)classIndex, (int*)indexSorted, (bool*)suppress_1d,
		(float*)outConf, (float*)outLoc, (int*)outClass, topK, batchSize, layerNum, keepTopK);

	//float scaleW = float(srcW * 1.0 / tarW);
	//float scaleH = float(srcH * 1.0 / tarH);

	//scaleAndClip << <20, 512 >> > ((float*)locAfterNms, (float*)outLoc, srcW, srcH, scaleW, scaleH, topK, batchSize, layerNum, keepTopK);

	hipFree(d_temp_storage);
	hipFree(d_temp);
	return STATUS_SUCCESS;
} 